#include "core/config/config.h"
#include "core/tool/logger.h"
#include "core/vulkan/vulkan_context.h"
#include "cuda_engine.h"
#include "function/global_context.h"
#include <cassert>

void CudaEngine::importExtBuffer(const ExtBufferDesc& buffer_desc)
{
    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    {
#ifdef _WIN64
        externalMemoryDesc.type                = hipExternalMemoryHandleTypeOpaqueWin32;
        externalMemoryDesc.handle.win32.handle = buffer_desc.handle; // File descriptor from Vulkan
#else
        externalMemoryDesc.type      = hipExternalMemoryHandleTypeOpaqueFd;
        externalMemoryDesc.handle.fd = buffer_desc.fd; // File descriptor from Vulkan
#endif

        externalMemoryDesc.size = buffer_desc.buffer_size;
    }
    hipExternalMemory_t ext_mem;
    hipImportExternalMemory(&ext_mem, &externalMemoryDesc);

    hipExternalMemoryBufferDesc bufferDesc = {};
    {
        bufferDesc.offset = 0;
        bufferDesc.size   = buffer_desc.buffer_size;
    }
    void* dev_ptr;
    hipExternalMemoryGetMappedBuffer(&dev_ptr, ext_mem, &bufferDesc);

    extBuffers[buffer_desc.name] = {
        ext_mem,
        dev_ptr,
        buffer_desc.buffer_size
    };
}

void CudaEngine::importExtImage(const ExtImageDesc& image_desc)
{
    static std::unordered_map<VkFormat, hipChannelFormatDesc> channel_map {
            { VK_FORMAT_R32_SFLOAT, { 32, 0, 0, 0, hipChannelFormatKindFloat } },
            { VK_FORMAT_S8_UINT,    { 8, 0, 0, 0, hipChannelFormatKindUnsigned } },
            { VK_FORMAT_R16G16B16A16_SFLOAT, { 16, 16, 16, 16, hipChannelFormatKindFloat } }
    };

    assert(image_desc.width * image_desc.height * image_desc.depth == image_desc.image_size / image_desc.element_size);
    assert(channel_map.contains(image_desc.format));

    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    {
#ifdef _WIN64
        externalMemoryDesc.type                = hipExternalMemoryHandleTypeOpaqueWin32;
        externalMemoryDesc.handle.win32.handle = image_desc.handle; // File descriptor from Vulkan
#else
        externalMemoryDesc.type      = hipExternalMemoryHandleTypeOpaqueFd;
        externalMemoryDesc.handle.fd = image_desc.fd; // File descriptor from Vulkan
#endif
    }
    externalMemoryDesc.size = image_desc.image_size;
    hipExternalMemory_t ext_mem;

    hipError_t res = hipImportExternalMemory(&ext_mem, &externalMemoryDesc);
    if (res != hipSuccess)
        ERROR_ALL("hipImportExternalMemory failed: " + std::string(hipGetErrorString(res)));

    hipExtent extent = make_hipExtent(image_desc.width, image_desc.height, image_desc.depth);
    cudaExternalMemoryMipmappedArrayDesc ext_mipmapped_arr_desc;
    {
        memset(&ext_mipmapped_arr_desc, 0, sizeof(ext_mipmapped_arr_desc));
        ext_mipmapped_arr_desc.offset     = 0;
        ext_mipmapped_arr_desc.formatDesc = channel_map[image_desc.format];
        ext_mipmapped_arr_desc.extent     = extent;
        ext_mipmapped_arr_desc.flags      = 0;
        ext_mipmapped_arr_desc.numLevels  = 1;
    }
    hipMipmappedArray_t mipmapped_arr;
    res = cudaExternalMemoryGetMappedMipmappedArray(&mipmapped_arr, ext_mem, &ext_mipmapped_arr_desc);
    if (res != hipSuccess)
        ERROR_ALL("cudaExternalMemoryGetMappedMipmappedArray failed: " + std::string(hipGetErrorString(res)));

    hipArray_t arr_0;
    hipGetMipmappedArrayLevel(&arr_0, mipmapped_arr, 0);

    hipResourceDesc res_desc = {};
    {
        res_desc.resType         = hipResourceTypeArray;
        res_desc.res.array.array = arr_0;
    }
    hipSurfaceObject_t surface_object;
    res = hipCreateSurfaceObject(&surface_object, &res_desc);
    if (res != hipSuccess)
        ERROR_ALL("hipCreateSurfaceObject failed: " +  std::string(hipGetErrorString(res)));

    extImages[image_desc.name] = {
        ext_mem,
        mipmapped_arr,
        surface_object,
        extent,
        image_desc.image_size,
        image_desc.element_size
    };
}

void CudaEngine::initExternalMem()
{
}

void CudaEngine::initSemaphore()
{
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type                = hipExternalSemaphoreHandleTypeOpaqueWin32;
    externalSemaphoreHandleDesc.handle.win32.handle = g_ctx->vk.cuUpdateSemaphoreHandle;
#else
    externalSemaphoreHandleDesc.type      = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = g_ctx->vk.cuUpdateSemaphoreFd;
#endif
    externalSemaphoreHandleDesc.flags = 0;
    hipImportExternalSemaphore(&cuUpdateSemaphore, &externalSemaphoreHandleDesc);

    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type                = hipExternalSemaphoreHandleTypeOpaqueWin32;
    externalSemaphoreHandleDesc.handle.win32.handle = g_ctx->vk.vkUpdateSemaphoreHandle;
#else
    externalSemaphoreHandleDesc.type      = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = g_ctx->vk.vkUpdateSemaphoreFd;
#endif
    externalSemaphoreHandleDesc.flags = 0;
    hipImportExternalSemaphore(&vkUpdateSemaphore, &externalSemaphoreHandleDesc);
}

void CudaEngine::init(Configuration& cfg, GlobalContext* g_ctx)
{
    this->g_ctx = g_ctx;

    hipStreamCreate(&streamToRun);
    initSemaphore();
    initExternalMem();
    signalSemaphore(cuUpdateSemaphore);
}

void CudaEngine::step()
{
    waitOnSemaphore(vkUpdateSemaphore);

    // TODO

    signalSemaphore(cuUpdateSemaphore);
}

void CudaEngine::sync()
{
    hipDeviceSynchronize();
}

void CudaEngine::cleanup()
{
    for (auto& p : extBuffers) {
        p.second.cleanup();
    }
    for (auto& p : extImages) {
        p.second.cleanup();
    }
    hipDestroyExternalSemaphore(vkUpdateSemaphore);
    hipDestroyExternalSemaphore(cuUpdateSemaphore);
}

void CudaEngine::waitOnSemaphore(hipExternalSemaphore_t& semaphore)
{
    hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
    memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
    extSemaphoreWaitParams.params.fence.value = 0;
    extSemaphoreWaitParams.flags              = 0;

    hipWaitExternalSemaphoresAsync(
        &semaphore, &extSemaphoreWaitParams, 1, streamToRun);
}

void CudaEngine::signalSemaphore(hipExternalSemaphore_t& semaphore)
{
    hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
    memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
    extSemaphoreSignalParams.params.fence.value = 0;
    extSemaphoreSignalParams.flags              = 0;

    hipSignalExternalSemaphoresAsync(
        &semaphore, &extSemaphoreSignalParams, 1, streamToRun);
}
